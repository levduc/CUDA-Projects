
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <math.h>
#include <algorithm>
#include <stdio.h>
#include <fcntl.h>
#include <time.h>
#define NS_PER_SEC (1000*1000*1000)
using namespace std;

int base[3][4];
int base7[3][7];
int tranposeBase7[7][3];
int base8[3][8];
int tranposeBase8[8][3];
int base11[3][11];
int base12[3][12];
int base13[3][13];
int base14[3][14];
int board7[7][7];
int board8[8][8];

inline unsigned long int monotonicTime(void)
{
  //const unsigned long int NS_PER_SEC = 1000 * 1000 * 1000;
  struct timespec now;
  clock_gettime(CLOCK_MONOTONIC, &now);
  return now.tv_sec * NS_PER_SEC + now.tv_nsec;
}

void loadData()
{
 //base 3*4
	base[0][0]=1;
	base[0][1]=4;
	base[0][2]=7;
	base[0][3]=10;
	base[1][0]=8;
	base[1][1]=11;
	base[1][2]=2;
	base[1][3]=5;
	base[2][0]=3;
	base[2][1]=6;
	base[2][2]=9;
	base[2][3]=12;

    //base 3*7
    	base7[0][0]=1;
	base7[0][1]=14;
	base7[0][2]=17;
	base7[0][3]=20;
	base7[0][4]=9;
	base7[0][5]=4;
	base7[0][6]=7;
	base7[1][0]=16;
	base7[1][1]=19;
	base7[1][2]=12;
	base7[1][3]=3;
	base7[1][4]=6;
	base7[1][5]=21;
	base7[1][6]=10;
	base7[2][0]=13;
	base7[2][1]=2;
	base7[2][2]=15;
	base7[2][3]=18;
	base7[2][4]=11;
	base7[2][5]=8;
	base7[2][6]=5;
    //Tranpose base 7
	for(int x = 0; x < 3; x++)
	  {
		for(int y = 0; y < 7 ; y++)
		{
		    	tranposeBase7[y][x] = base7[x][y];                
	    	}
	  }
    //base 3*8
    	base8[0][0]=1;
	base8[0][1]=16;
	base8[0][2]=3;
	base8[0][3]=22;
	base8[0][4]=19;
	base8[0][5]=12;
	base8[0][6]=7;
	base8[0][7]=10;
	base8[1][0]=4;
	base8[1][1]=21;
	base8[1][2]=18;
	base8[1][3]=15;
	base8[1][4]=6;
	base8[1][5]=9;
	base8[1][6]=24;
	base8[1][7]=13;
	base8[2][0]=17;
	base8[2][1]=2;
	base8[2][2]=5;
	base8[2][3]=20;
	base8[2][4]=23;
	base8[2][5]=14;
	base8[2][6]=11;
	base8[2][7]=8;
    //Tranpose base 8
	for(int x = 0; x < 3; x++)
	  {
		for(int y = 0; y < 8 ; y++)
		{
		    	tranposeBase8[y][x] = base8[x][y];                
	    	}
	  }	
	
    //base 3*11
    for(int x = 0; x < 3; x++)
    {
        for(int y = 0; y < 11 ; y++)
        {
            if(y < 4)
            	base11[x][y] = base[x][y];            
            else
            	base11[x][y] = base7[x][y-4]+3*4;     
    	}// end of inner loop
    }
    //base 3*12
    for(int x = 0; x < 3; x++)
    {
        for(int y =0; y<12; y++)
        {
            if(y<4)
            base12[x][y] = base[x][y];
            else
            base12[x][y] = base8[x][y-4]+3*4;
	}
    }
    //board 3*13
    	base13[0][0]=1;
	base13[0][1]=4;
	base13[0][2]=13;
	base13[0][3]=16;
	base13[0][4]=21;
	base13[0][5]=8;
	base13[0][6]=23;
	base13[0][7]=18;
	base13[0][8]=35;
	base13[0][9]=38;
	base13[0][10]=27;
	base13[0][11]=32;
	base13[0][12]=29;
	base13[1][0]=12;
	base13[1][1]=15;
	base13[1][2]=6;
	base13[1][3]=3;
	base13[1][4]=10;
	base13[1][5]=17;
	base13[1][6]=20;
	base13[1][7]=37;
	base13[1][8]=24;
	base13[1][9]=33;
	base13[1][10]=30;
	base13[1][11]=39;
	base13[1][12]=26;
	base13[2][0]=5;
	base13[2][1]=2;
	base13[2][2]=11;
	base13[2][3]=14;
	base13[2][4]=7;
	base13[2][5]=22;
	base13[2][6]=9;
	base13[2][7]=34;
	base13[2][8]=19;
	base13[2][9]=36;
	base13[2][10]=25;
	base13[2][11]=28;
	base13[2][12]=31;
    //base 3*14
    for(int x = 0; x < 3; x++)
    {	
       for(int y =0; y<14; y++)
        {
            if(y < 7)
                base14[x][y] = base7[x][y];
            else
               base14[x][y] = base7[x][y-7]+3*7;         
     	}
    }
    // load board 7
   	board7[0][0] = 1;
	board7[0][1] = 26;
	board7[0][2] = 11;
	board7[0][3] = 46;
	board7[0][4] = 29;
	board7[0][5] = 24;
	board7[0][6] = 9;
	board7[1][0] = 12;
	board7[1][1] = 45;
	board7[1][2] = 28;
	board7[1][3] = 25;
	board7[1][4] = 10;
	board7[1][5] = 47;
	board7[1][6] = 30;
	board7[2][0] = 27;
	board7[2][1] = 2;
	board7[2][2] = 35;
	board7[2][3] = 44;
	board7[2][4] = 49;
	board7[2][5] = 8;
	board7[2][6] = 23;
	board7[3][0] = 40;
	board7[3][1] = 13;
	board7[3][2] = 42;
	board7[3][3] = 19;
	board7[3][4] = 36;
	board7[3][5] = 31;
	board7[3][6] = 48;
	board7[4][0] = 3;
	board7[4][1] = 16;
	board7[4][2] = 39;
	board7[4][3] = 34;
	board7[4][4] = 43;
	board7[4][5] = 22;
	board7[4][6] = 7;
	board7[5][0] = 14;
	board7[5][1] = 41;
	board7[5][2] = 18;
	board7[5][3] = 5;
	board7[5][4] = 20;
	board7[5][5] = 37;
	board7[5][6] = 32;
	board7[6][0] = 17;
	board7[6][1] = 4;
	board7[6][2] = 15;
	board7[6][3] = 38;
	board7[6][4] = 33;
	board7[6][5] = 6;
	board7[6][6] = 21;
   //Board 8
	 board8[0][0] = 1;
	board8[0][1] = 46;
	board8[0][2] = 15;
	board8[0][3] = 24;
	board8[0][4] = 59;
	board8[0][5] = 28;
	board8[0][6] = 13;
	board8[0][7] = 26;
	board8[1][0] = 16;
	board8[1][1] = 23;
	board8[1][2] = 58;
	board8[1][3] = 51;
	board8[1][4] = 14;
	board8[1][5] = 25;
	board8[1][6] = 64;
	board8[1][7] = 29;
	board8[2][0] = 47;
	board8[2][1] = 2;
	board8[2][2] = 45;
	board8[2][3] = 54;
	board8[2][4] = 63;
	board8[2][5] = 60;
	board8[2][6] = 27;
	board8[2][7] = 12;
	board8[3][0] = 22;
	board8[3][1] = 17;
	board8[3][2] = 52;
	board8[3][3] = 57;
	board8[3][4] = 50;
	board8[3][5] = 55;
	board8[3][6] = 30;
	board8[3][7] = 61;
	board8[4][0] = 3;
	board8[4][1] = 48;
	board8[4][2] = 21;
	board8[4][3] = 44;
	board8[4][4] = 53;
	board8[4][5] = 62;
	board8[4][6] = 11;
	board8[4][7] = 34;
	board8[5][0] = 18;
	board8[5][1] = 39;
	board8[5][2] = 42;
	board8[5][3] = 49;
	board8[5][4] = 56;
	board8[5][5] = 33;
	board8[5][6] = 8;
	board8[5][7] = 31;
	board8[6][0] = 41;
	board8[6][1] = 4;
	board8[6][2] = 37;
	board8[6][3] = 20;
	board8[6][4] = 43;
	board8[6][5] = 6;
	board8[6][6] = 35;
	board8[6][7] = 10;
	board8[7][0] = 38;
	board8[7][1] = 19;
	board8[7][2] = 40;
	board8[7][3] = 5;
	board8[7][4] = 36;
	board8[7][5] = 9;
	board8[7][6] = 32;
	board8[7][7] = 7;

}

int blockOfFour(int n) // getting num blocks of four in each stripe.
{
    if(n < 11)
    {
	return 0;
    }
    else
    {
        int num = 0;
        switch(n%4)
        {
	    case 0:
	       num = (n-8);
	       break;
	    case 1:
	       num = (n-13);
	       break;
	    case 2:
	       num = (n-14);
	       break;
	    case 3:
	       num = (n-7);
	       break;
        }
	return num;
    }		
}


void solveBoard(int n)
{
    int board[n][n]; // initialize the board
    for (int i = 0; i < n; i++)
       for (int j = 0; j < n; j++)
            board[i][j] = 0;
    int BaseOfFour = blockOfFour(n)/4; // number blocks of 3*4
    
    switch(n % 3)
    {	
	case 0: // for all board size that is divisibe by 3
	   for (int x = 0; x < 3; x++)
	   {
	       for(int y = 0; y < n; y++)
	       {
		    if(y < blockOfFour(n))
		    {
			int temp = y/4;	
			for(int i = 0; i < n; i+= 6) // parrallel here parrallel here i+6 to get stripe without flipping order
			{
			    int stride = i/3;		    	
			    board[x+i][y] = base[x][y%4]+ temp*12 + 3*n*stride;
			    if(x+3+i < n)
			    	board[x+3+i][n-y-1] = base[x][y%4]+ temp*12 + 3*n*(stride+1);
			}
		    }
		    else
		    {
			for(int i = 0 ; i < n; i+= 6) // parallel here
			{
			    int stride = i/3;
			    if(n % 4 == 0)
			    {
			    	board[x+i][y] = base8[x][y-blockOfFour(n)]+BaseOfFour*12 + 3*n*stride;
				board[x+3+i][n-y-1] = base8[x][y-blockOfFour(n)]+ BaseOfFour*12 + 3*n*(stride+1); //using base 8
			    }
 			    if(n % 4 == 1)
			    {
			    	board[x+i][y] = base13[x][y-blockOfFour(n)]+ BaseOfFour * 12 + 3*n*stride;
				if(x+3+i < n)
				   board[x+3+i][n-y-1] = base13[x][y-blockOfFour(n)]+ BaseOfFour * 12 + 3*n*(stride+1); //using base 13
			    }
			    if(n % 4 == 2)
			    {
			    	board[x+i][y] = base14[x][y-blockOfFour(n)]+ BaseOfFour * 12 + 3*n*stride;
				if(x+3+i < n)
				   board[x+3+i][n-y-1] = base14[x][y-blockOfFour(n)]+ BaseOfFour * 12 + 3*n*(stride+1); // using base 14
			    }
			    if(n % 4 == 3)
			    {
			    	board[x+i][y] = base7[x][y-blockOfFour(n)]+ BaseOfFour * 12 + 3*n*stride;
				if(x+3+i < n)
				   board[x+3+i][n-y-1] = base7[x][y-blockOfFour(n)]+ BaseOfFour * 12 + 3*n*(stride+1); // using base 7
			    }
			}
		    }
	       }   
	   }
	   break; //  end of first case
	case 1:
	   for (int x = 0; x < 3; x++)
	   {
	       for(int y = 0; y < n; y++)
	       {
		    if(y < blockOfFour(n))
		    {
			int temp = y/4; // temp get index of blockOf4
			for(int i = 0; i < n-7; i+= 6) // 
			{
			    int stride = i/3; 	    	
			    board[x+i][y] = base[x][y%4]+ temp*12 + 3*n*stride; 
			    if(x+3+i < n-7)
			    	board[x+3+i][n-y-1] = base[x][y%4]+ temp*12 + 3*n*(stride+1);
			}
		    }// end of if
		    else
		    {
			for(int i = 0 ; i < n-7; i+= 6) // parallel here
			{
			    int stride = i/3;
			    if(n % 4 == 0)
			    {
			    	board[x+i][y] = base8[x][y-blockOfFour(n)]+BaseOfFour*12 + 3*n*stride;
				if(x+3+i < n-7)// Don't want to get in the 7 stride
				board[x+3+i][n-y-1] = base8[x][y-blockOfFour(n)]+ BaseOfFour*12 + 3*n*(stride+1);
			    }
 			    if(n % 4 == 1)
			    {
			    	board[x+i][y] = base13[x][y-blockOfFour(n)]+ BaseOfFour * 12 + 3*n*stride;
				if(x+3+i < n-7) // Don't want to get in the 7 stride
				   board[x+3+i][n-y-1] = base13[x][y-blockOfFour(n)]+ BaseOfFour * 12 + 3*n*(stride+1);
			    }
			    if(n % 4 == 2)
			    {
			    	board[x+i][y] = base14[x][y-blockOfFour(n)]+ BaseOfFour * 12 + 3*n*stride;
				if(x+3+i < n-7) // Don't want to get in the 7 stride
				   board[x+3+i][n-y-1] = base14[x][y-blockOfFour(n)]+ BaseOfFour * 12 + 3*n*(stride+1);
			    }
			    if(n % 4 == 3)
			    {
			    	board[x+i][y] = base7[x][y-blockOfFour(n)]+ BaseOfFour * 12 + 3*n*stride;
				if(x+3+i < n-7) // Don't want to get in the 7 stride
				   board[x+3+i][n-y-1] = base7[x][y-blockOfFour(n)]+ BaseOfFour * 12 + 3*n*(stride+1);
			    }
			}
		    }// end of else
	       }// end of for y   
	   }// end of big 4
	   //Handling 7*3(k-1) here 2 case 
	   if(n%2 != 0)
	   {
		   for(int x = 0; x < 7; x++)
		   {
			for(int y = 0; y < n-7 ; y++) // NOTE !!!!
			{
			    if(y%6 == 0) //0,6,12....
			    {
				int temp = y/6;
			    	board[n+x-7][y] = tranposeBase7[x][y%3]+ 2*temp*21 + (n-7)*n; // minus 7 because we want to start at line n-7
				board[n+x-7][y+1] = tranposeBase7[x][y%3+1]+ 2*temp*21 + (n-7)*n;
				board[n+x-7][y+2] = tranposeBase7[x][y%3+2]+ 2*temp*21 + (n-7)*n;
			    }
			    else if( y%3 == 0) //3,9,15,21...
			    {
				int temp = y/6;
			    	board[n-x-1][y] = tranposeBase7[x][y%3]+ 21 + 2*temp*21 + (n-7)*n;
				board[n-x-1][y+1] = tranposeBase7[x][y%3+1]+ 21 + 2*temp*21+ (n-7)*n;
				board[n-x-1][y+2] = tranposeBase7[x][y%3+2]+ 21 + 2*temp*21+ (n-7)*n;
			    }
			}
		   }
		   //handling 7*7 chessboard here
		   for(int x = 0; x < 7; x++)
		   {
			for(int y = 0; y < 7 ; y++)
			{
			    board[n+x-7][n+y-7] = board7[x][y] + (n*n-49);
			}
		   }
           } else
		{
		for(int x = 0; x < 7; x++)
		   {
			for(int y = n; y > 7 ; y--) // NOTE !!!!
			{
			    if((n-y)%6 == 0) // n-y because we start from ending
			    {
				int temp = (n-y)/6;
			    	board[n+x-7][y-1] = tranposeBase7[x][(n-y)%3] + 2*temp*21 + (n-7)*n;
				board[n+x-7][y-2] = tranposeBase7[x][(n-y)%3+1] + 2*temp*21 + (n-7)*n; //+ 2*temp*21 + (n-7)*n
				board[n+x-7][y-3] = tranposeBase7[x][(n-y)%3+2] + 2*temp*21 + (n-7)*n; 
			    }
			    else if((n-y)%3 == 0) // n-y because we start from ending
			    {
				int temp = (n-y)/6;
			    	board[n-x-1][y-1] = tranposeBase7[x][(n-y)%3] + 21 +  2*temp*21 + (n-7)*n;
				board[n-x-1][y-2] = tranposeBase7[x][(n-y)%3+1]+ 21 + 2*temp*21 + (n-7)*n;
				board[n-x-1][y-3] = tranposeBase7[x][(n-y)%3+2]+ 21 + 2*temp*21 + (n-7)*n;
			    }
			}
		   }
		   //handling 7*7 chessboard here
		   for(int x = 6; x >= 0; x--)
		   {
			for(int y = 6; y >=0 ; y--)
			{
			    board[n+x-7][y] = board7[6-x][6-y] + (n*n-49);
			    //cout << "testing" << endl;
			}
		   }
	   }// end of else

	  
	
	break;
      case 2:
	 for (int x = 0; x < 3; x++)
	   {
	       for(int y = 0; y < n; y++)
	       {
		    if(y < blockOfFour(n))
		    {
			int temp = y/4;	
			for(int i = 0; i < n-8; i+= 6) 
			{
			    int stride = i/3; 	    	
			    board[x+i][y] = base[x][y%4]+ temp*12 + 3*n*stride;
			    if(x+3+i < n-8)
			    	board[x+3+i][n-y-1] = base[x][y%4]+ temp*12 + 3*n*(stride+1);
			}
		    }// end of if
		    else
		    {
			for(int i = 0 ; i < n-8; i+= 6)
			{
			    int stride = i/3;
			    if(n % 4 == 0)
			    {
			    	board[x+i][y] = base8[x][y-blockOfFour(n)]+BaseOfFour*12 + 3*n*stride;
				if(x+3+i < n-8)// Don't want to get in the 8 stride
				board[x+3+i][n-y-1] = base8[x][y-blockOfFour(n)]+ BaseOfFour*12 + 3*n*(stride+1);
			    }
 			    if(n % 4 == 1)
			    {
			    	board[x+i][y] = base13[x][y-blockOfFour(n)]+ BaseOfFour * 12 + 3*n*stride;
				if(x+3+i < n-8) // Don't want to get in the 8 stride
				   board[x+3+i][n-y-1] = base13[x][y-blockOfFour(n)]+ BaseOfFour * 12 + 3*n*(stride+1);
			    }
			    if(n % 4 == 2)
			    {
			    	board[x+i][y] = base14[x][y-blockOfFour(n)]+ BaseOfFour * 12 + 3*n*stride;
				if(x+3+i < n-8) // Don't want to get in the 8 stride
				   board[x+3+i][n-y-1] = base14[x][y-blockOfFour(n)]+ BaseOfFour * 12 + 3*n*(stride+1);
			    }
			    if(n % 4 == 3)
			    {
			    	board[x+i][y] = base7[x][y-blockOfFour(n)]+ BaseOfFour * 12 + 3*n*stride;
				if(x+3+i < n-8) // Don't want to get in the 8 stride
				   board[x+3+i][n-y-1] = base7[x][y-blockOfFour(n)]+ BaseOfFour * 12 + 3*n*(stride+1);
			    }
			}
		    }// end of else
	       }// end of for y   
	   }// end of big 4
	// Handling 8*n stride here
	 if(n%2 == 0)
	   {
		   for(int x = 0; x < 8; x++)
		   {
			for(int y = 0; y < n-8 ; y++) // NOTE !!!!
			{
			    if(y%6 == 0) //0,6,12....
			    {
				int temp = y/6;
			    	board[n+x-8][y] = tranposeBase8[x][y%3]+ 2*temp*24 + (n-8)*n; // minus 8 because we want to start at line n-8
				board[n+x-8][y+1] = tranposeBase8[x][y%3+1]+ 2*temp*24 + (n-8)*n;
				board[n+x-8][y+2] = tranposeBase8[x][y%3+2]+ 2*temp*24 + (n-8)*n;
			    }
			    else if( y%3 == 0) //3,9,15,21...
			    {
				int temp = y/6;
			    	board[n-x-1][y] = tranposeBase8[x][y%3]+ 24 + 2*temp*24 + (n-8)*n;
				board[n-x-1][y+1] = tranposeBase8[x][y%3+1]+ 24 + 2*temp*24+ (n-8)*n;
				board[n-x-1][y+2] = tranposeBase8[x][y%3+2]+ 24 + 2*temp*24+ (n-8)*n;
			    }
			}
		   }
		   //handling 8*8 chessboard here
		   for(int x = 0; x < 8; x++)
		   {
			for(int y = 0; y < 8 ; y++)
			{
			    board[n+x-8][n+y-8] = board8[x][y] + (n*n-64);
			}
		   }
           } else
		{
		for(int x = 0; x < 8; x++)
		   {
			for(int y = n; y > 8 ; y--) // NOTE !!!!
			{
			    if((n-y)%6 == 0) // n-y because we start from ending
			    {
				int temp = (n-y)/6; // getting index of block 8*3
			    	board[n+x-8][y-1] = tranposeBase8[x][(n-y)%3] + 2*temp*24 + (n-8)*n;
				board[n+x-8][y-2] = tranposeBase8[x][(n-y)%3+1] + 2*temp*24 + (n-8)*n;
				board[n+x-8][y-3] = tranposeBase8[x][(n-y)%3+2] + 2*temp*24 + (n-8)*n; 
			    }
			    else if((n-y)%3 == 0) // n-y because we start from ending
			    {
				int temp = (n-y)/6; // getting index of block 8*3
			    	board[n-x-1][y-1] = tranposeBase8[x][(n-y)%3] + 24 +  2*temp*24 + (n-8)*n; // note 24 = 3*8 
				board[n-x-1][y-2] = tranposeBase8[x][(n-y)%3+1]+ 24 + 2*temp*24 + (n-8)*n;
				board[n-x-1][y-3] = tranposeBase8[x][(n-y)%3+2]+ 24 + 2*temp*24 + (n-8)*n;
			    }
			}
		   }
		   //handling 7*7 chessboard here
		   for(int x = 7; x >= 0; x--)
		   {
			for(int y = 7; y >=0 ; y--)
			{
			    board[n+x-7][y] = board8[7-x][7-y] + (n*n-64);
			    //cout << "testing" << endl;
			}
		   }
	   }// end of else
    } // end of switch


    // print out the thingy
    
    /*for (int x = 0; x < n; x++) {
        for (int y = 0; y < n; y++)
            cout << board[x][y]<< "\t";
        cout << endl;
    }*/
    
}

int main()
{
    loadData();
    int n;
    cout << "Enter size of board:";
    cin >> n;
    unsigned long int cpuTime = monotonicTime();
    solveBoard(n);
    cpuTime = monotonicTime() - cpuTime;  
    fprintf(stderr, "Time to perform operation on CPU = %ld ns\n", cpuTime);
    /*for(int x = 0; x < 3; x++)
    {
	for(int y = 0; y < 8 ; y++)
	    	cout << base8[x][y] << "\t";                
	cout << endl;
    }
    for(int x = 0; x < 8; x++)
    {
	for(int y = 0; y < 3 ; y++)
	    	cout << tranposeBase8[x][y] << "\t";   
	cout << endl;             
    }*/
    return 0;
}
