
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <math.h>
#include <algorithm>
#include <stdio.h>
#include <fcntl.h>
#include <time.h>
#define NS_PER_SEC (1000*1000*1000)
using namespace std;

int base[12];
int base7[21];
int base8[24];
int base11[33];
int base12[36];
int base13[39];
int base14[42];

inline unsigned long int monotonicTime(void)
{
  //const unsigned long int NS_PER_SEC = 1000 * 1000 * 1000;
  struct timespec now;
  clock_gettime(CLOCK_MONOTONIC, &now);
  return now.tv_sec * NS_PER_SEC + now.tv_nsec;
}

void loadData()
{
 //base 3*4
	base[0]=1;
	base[3]=4;
	base[6]=7;
	base[9]=10;
	base[1]=8;
	base[4]=11;
	base[7]=2;
	base[10]=5;
	base[2]=3;
	base[5]=6;
	base[8]=9;
	base[11]=12;

    //base 3*7
    	base7[0]=1;
	base7[3]=14;
	base7[6]=17;
	base7[9]=20;
	base7[12]=9;
	base7[15]=4;
	base7[18]=7;
	base7[1]=16;
	base7[4]=19;
	base7[7]=12;
	base7[10]=3;
	base7[13]=6;
	base7[16]=21;
	base7[19]=10;
	base7[2]=13;
	base7[5]=2;
	base7[8]=15;
	base7[11]=18;
	base7[14]=11;
	base7[17]=8;
	base7[20]=5;
    //base 3*8
    	base8[0]=1;
	base8[3]=16;
	base8[6]=3;
	base8[9]=22;
	base8[12]=19;
	base8[15]=12;
	base8[18]=7;
	base8[21]=10;
	base8[1]=4;
	base8[4]=21;
	base8[7]=18;
	base8[10]=15;
	base8[13]=6;
	base8[16]=9;
	base8[19]=24;
	base8[22]=13;
	base8[2]=17;
	base8[5]=2;
	base8[8]=5;
	base8[11]=20;
	base8[14]=23;
	base8[17]=14;
	base8[20]=11;
	base8[23]=8;
    //base 3*11
    for(int x = 0; x < 33; x++)
    {
            if(x < 12)
            base11[x] = base[x];            
            else
            base11[x] = base7[x-12]+3*4;     
    }
    //base 3*12
    for(int x = 0; x < 36; x++)
    {
            if(x<12)
            base12[x] = base[x];
            else
            base12[x] = base8[x-12]+3*4;
    }
    //board 3*13
    	base13[0]=1;
	base13[3]=4;
	base13[6]=13;
	base13[9]=16;
	base13[12]=21;
	base13[15]=8;
	base13[18]=23;
	base13[21]=18;
	base13[24]=35;
	base13[27]=38;
	base13[30]=27;
	base13[33]=32;
	base13[36]=29;
	base13[1]=12;
	base13[4]=15;
	base13[7]=6;
	base13[10]=3;
	base13[13]=10;
	base13[16]=17;
	base13[19]=20;
	base13[22]=37;
	base13[25]=24;
	base13[28]=33;
	base13[31]=30;
	base13[34]=39;
	base13[37]=26;
	base13[2]=5;
	base13[5]=2;
	base13[8]=11;
	base13[11]=14;
	base13[14]=7;
	base13[17]=22;
	base13[20]=9;
	base13[23]=34;
	base13[26]=19;
	base13[29]=36;
	base13[32]=25;
	base13[35]=28;
	base13[38]=31;
    //base 3*14
    for(int x = 0; x < 42; x++)
    {	
            if(x < 21)
                base14[x] = base7[x];
            else
               base14[x] = base7[x-21]+3*7;         
    }
}

int blockOfFour(int n) // getting num blocks of four in each stripe.
{
    if(n < 11)
    {
	return 0;
    }
    else
    {
        int num = 0;
        switch(n%4)
        {
	    case 0:
	       num = (n-8);
	       break;
	    case 1:
	       num = (n-13);
	       break;
	    case 2:
	       num = (n-14);
	       break;
	    case 3:
	       num = (n-7);
	       break;
        }
	return num;
    }		
}
__device__ int gpuBlockOfFour(int n)
{
    if(n < 11)
    {
	return 0;
    }
    else
    {
        int num = 0;
        switch(n%4)
        {
	    case 0:
	       num = (n-8);
	       break;
	    case 1:
	       num = (n-13);
	       break;
	    case 2:
	       num = (n-14);
	       break;
	    case 3:
	       num = (n-7);
	       break;
        }
	return num;
    }		
}

__global__ void solveBoard(int* base, int* base7, int* base8, int* base11, int* base12, int* base13, int* base14, int* board, int* runTime, int n)
{
    clock_t start_time = clock();
    switch(n % 3)
    {	
	case 0: // for all board size that is divisibe by 3
	int BaseOfFour = gpuBlockOfFour(n)/4;
	int blockOfFour = gpuBlockOfFour(n);
	   for (int x = 0; x < 3; x++)
	   {
	       for(int y = 0; y < n; y++)
	       {
		    if(y < gpuBlockOfFour(n))
		    {
			int temp = y/4;	
			//for(int i = 0; i < n; i+= 6) // parrallel here if(threadIdx.x%2 == 0) i = threadIdx*6
			//{
			if(threadIdx.x % 2 == 0)
			{
			    int i =  threadIdx.x*3;
			    //int stride = threadIdx.x;			    	
			    board[(x+i)+n*y] = base[x+(y%4)*3]+ temp*12 + 3*n*threadIdx.x;
			    if(x+3+i < n)
			    	board[(x+3+i)+n*(n-y-1)] = base[x+(y%4)*3]+ temp*12 + 3*n*(threadIdx.x+1);
			}
			//}
		    }
		    else
		    {
			//for(int i = 0 ; i < n; i+= 6) // parallel here
			//{
			if(threadIdx.x % 2 == 0)
			{
			    int i =  threadIdx.x*3;
			    //int stride = i/3;
			    if(n % 4 == 0)
			    {
			    	board[(x+i)+n*y] = base8[x+(y-blockOfFour)*3]+BaseOfFour*12 + 3*n*threadIdx.x;
				board[(x+3+i)+n*(n-y-1)] = base8[x+(y-blockOfFour)*3]+ BaseOfFour*12 + 3*n*(threadIdx.x+1);
			    }
 			    if(n % 4 == 1)
			    {
			    	board[(x+i)+n*y] = base13[x+(y-blockOfFour)*3]+ BaseOfFour * 12 + 3*n*threadIdx.x;
				if(x+3+i < n)
				   board[(x+3+i)+ n*(n-y-1)] = base13[x+(y-blockOfFour)*3]+ BaseOfFour * 12 + 3*n*(threadIdx.x+1);
			    }
			    if(n % 4 == 2)
			    {
			    	board[(x+i)+ n*y] = base14[x+(y-blockOfFour)*3]+ BaseOfFour * 12 + 3*n*threadIdx.x;
				if(x+3+i < n)
				   board[(x+3+i)+n*(n-y-1)] = base14[x+(y-blockOfFour)*3]+ BaseOfFour * 12 + 3*n*(threadIdx.x+1);
			    }
			    if(n % 4 == 3)
			    {
			    	board[(x+i)+n*y] = base7[x+(y-blockOfFour)*3]+ BaseOfFour * 12 + 3*n*threadIdx.x;
				if(x+3+i < n)
				   board[(x+3+i)+n*(n-y-1)] = base7[x+(y-blockOfFour)*3]+ BaseOfFour * 12 + 3*n*(threadIdx.x+1);
			    }
			}
		    }
	       }   
	   }
	   break; 
    }
    clock_t stop_time = clock();
    runTime[threadIdx.x] =(int) (stop_time - start_time);
    /*for (int x = 0; x < n; x++) {
        for (int y = 0; y < n; y++)
            cout << board[x+n*y]<< "\t";
        cout << endl;
    }*/
}

int main()
{
    loadData();
    int n;
    cout << "Enter size of board:";
    cin >> n;
    int board[n*n];
    int Time[n/3];
    unsigned long int gpuTime = monotonicTime();
    // Declare gpuBase
    int* gpuBase;
    int* gpuBase7;
    int* gpuBase8;
    int* gpuBase11;
    int* gpuBase12;
    int* gpuBase13;
    int* gpuBase14;
    int* gpuBoard;
    int* runTime;
    //Allocate
    hipMalloc(&gpuBase, 12*sizeof(int)); 
    hipMalloc(&gpuBase7, 21*sizeof(int));
    hipMalloc(&gpuBase8, 24*sizeof(int)); 
    hipMalloc(&gpuBase11, 33*sizeof(int));
    hipMalloc(&gpuBase12, 36*sizeof(int)); 
    hipMalloc(&gpuBase13, 39*sizeof(int));
    hipMalloc(&gpuBase14, 42*sizeof(int)); 
    hipMalloc(&gpuBoard, n*n*sizeof(int));
    hipMalloc(&runTime, n*sizeof(int)); //  getting runTime 
    //Copy data
    hipMemcpy(gpuBase, base,  12*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gpuBase7, base7,  21*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gpuBase8, base8,  24*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gpuBase11, base11,  33*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gpuBase12, base12,  36*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gpuBase13, base13,  39*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gpuBase14, base14,  42*sizeof(int), hipMemcpyHostToDevice);
    // Calculate threads needed
    int num_threads = n/3; // we want to handle the last thing in stripe
    // Call kernel
    solveBoard<<<1,num_threads>>>(gpuBase,gpuBase7,gpuBase8,gpuBase11,gpuBase12,gpuBase13,gpuBase14,gpuBoard,runTime, n);
    // copy to out from device to host
    hipMemcpy(board, gpuBoard, n*n* sizeof(int) , hipMemcpyDeviceToHost); 
    hipMemcpy(Time, runTime, (n/3)* sizeof(int) , hipMemcpyDeviceToHost); 	
    gpuTime = monotonicTime() - gpuTime;  
    fprintf(stderr, "Time to perform operation on CPU = %ld ns\n", Time[n/3-1]);

    /*for (int x = 0; x < n; x++) {
        for (int y = 0; y < n; y++)
            cout << board[x+n*y]<< "\t";
        cout << endl;
    }
    
    /*for(int x = 0; x < 42; x++)
    {         
	cout << base14[x] << endl;
    }*/
    return 0;
}
