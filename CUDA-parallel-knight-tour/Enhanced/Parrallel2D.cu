
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <math.h>
#include <algorithm>
using namespace std;

int base[3][4];
int base7[3][7];
int tranposeBase7[7][3];
int base8[3][8];
int base11[3][11];
int base12[3][12];
int base13[3][13];
int base14[3][14];
int board7[7][7];
int board8[8][8];
void loadData()
{
 //base 3*4
	base[0][0]=1;
	base[0][1]=4;
	base[0][2]=7;
	base[0][3]=10;
	base[1][0]=8;
	base[1][1]=11;
	base[1][2]=2;
	base[1][3]=5;
	base[2][0]=3;
	base[2][1]=6;
	base[2][2]=9;
	base[2][3]=12;

    //base 3*7
    	base7[0][0]=1;
	base7[0][1]=14;
	base7[0][2]=17;
	base7[0][3]=20;
	base7[0][4]=9;
	base7[0][5]=4;
	base7[0][6]=7;
	base7[1][0]=16;
	base7[1][1]=19;
	base7[1][2]=12;
	base7[1][3]=3;
	base7[1][4]=6;
	base7[1][5]=21;
	base7[1][6]=10;
	base7[2][0]=13;
	base7[2][1]=2;
	base7[2][2]=15;
	base7[2][3]=18;
	base7[2][4]=11;
	base7[2][5]=8;
	base7[2][6]=5;
    //Tranpose base 7
	for(int x = 0; x < 3; x++)
	  {
		for(int y = 0; y < 7 ; y++)
		{
		    	tranposeBase7[y][x] = base7[x][y];                
	    	}
	  }
    //base 3*8
    	base8[0][0]=1;
	base8[0][1]=16;
	base8[0][2]=3;
	base8[0][3]=22;
	base8[0][4]=19;
	base8[0][5]=12;
	base8[0][6]=7;
	base8[0][7]=10;
	base8[1][0]=4;
	base8[1][1]=21;
	base8[1][2]=18;
	base8[1][3]=15;
	base8[1][4]=6;
	base8[1][5]=9;
	base8[1][6]=24;
	base8[1][7]=13;
	base8[2][0]=17;
	base8[2][1]=2;
	base8[2][2]=5;
	base8[2][3]=20;
	base8[2][4]=23;
	base8[2][5]=14;
	base8[2][6]=11;
	base8[2][7]=8;
    //base 3*11
    for(int x = 0; x < 3; x++)
    {
        for(int y = 0; y < 11 ; y++)
        {
            if(y < 4)
            	base11[x][y] = base[x][y];            
            else
            	base11[x][y] = base7[x][y-4]+3*4;     
    	}// end of inner loop
    }
    //base 3*12
    for(int x = 0; x < 3; x++)
    {
        for(int y =0; y<12; y++)
        {
            if(y<4)
            base12[x][y] = base[x][y];
            else
            base12[x][y] = base8[x][y-4]+3*4;
	}
    }
    //board 3*13
    	base13[0][0]=1;
	base13[0][1]=4;
	base13[0][2]=13;
	base13[0][3]=16;
	base13[0][4]=21;
	base13[0][5]=8;
	base13[0][6]=23;
	base13[0][7]=18;
	base13[0][8]=35;
	base13[0][9]=38;
	base13[0][10]=27;
	base13[0][11]=32;
	base13[0][12]=29;
	base13[1][0]=12;
	base13[1][1]=15;
	base13[1][2]=6;
	base13[1][3]=3;
	base13[1][4]=10;
	base13[1][5]=17;
	base13[1][6]=20;
	base13[1][7]=37;
	base13[1][8]=24;
	base13[1][9]=33;
	base13[1][10]=30;
	base13[1][11]=39;
	base13[1][12]=26;
	base13[2][0]=5;
	base13[2][1]=2;
	base13[2][2]=11;
	base13[2][3]=14;
	base13[2][4]=7;
	base13[2][5]=22;
	base13[2][6]=9;
	base13[2][7]=34;
	base13[2][8]=19;
	base13[2][9]=36;
	base13[2][10]=25;
	base13[2][11]=28;
	base13[2][12]=31;
    //base 3*14
    for(int x = 0; x < 3; x++)
    {	
       for(int y =0; y<14; y++)
        {
            if(y < 7)
                base14[x][y] = base7[x][y];
            else
               base14[x][y] = base7[x][y-7]+3*7;         
     	}
    }
}

int blockOfFour(int n) // getting num blocks of four in each stripe.
{
    if(n < 11)
    {
	return 0;
    }
    else
    {
        int num = 0;
        switch(n%4)
        {
	    case 0:
	       num = (n-8);
	       break;
	    case 1:
	       num = (n-13);
	       break;
	    case 2:
	       num = (n-14);
	       break;
	    case 3:
	       num = (n-7);
	       break;
        }
	return num;
    }		
}

int** create2DArray(int n) 
    {
      int** array2D = 0;
      //transpose 3x7 to 7*3
      
      array2D = new int*[7]; // height is 7

      for (int h = 0; h < 7; h++)
      {
            array2D[h] = new int[n-7];
	    //initialize array
            for (int w = 0; w < (n-7); w++)
            {
                 //if(w < ) 
		 //array2D[h][w] = w + width * h;
            }
      }
      return array2D;
}

void solveBoard(int n)
{
    int board[n][n]; // initialize the board
    for (int i = 0; i < n; i++)
       for (int j = 0; j < n; j++)
            board[i][j] = 0;
    int BaseOfFour = blockOfFour(n)/4; // number blocks of 3*4
    
    switch(n % 3)
    {	
	case 0: // for all board size that is divisibe by 3
	   for (int x = 0; x < 3; x++)
	   {
	       for(int y = 0; y < n; y++)
	       {
		    if(y < blockOfFour(n))
		    {
			int temp = y/4;	
			for(int i = 0; i < n; i+= 6) // parrallel here parrallel here i+6 to get stripe without flipping order
			{
			    int stride = i/3;		    	
			    board[x+i][y] = base[x][y%4]+ temp*12 + 3*n*stride;
			    if(x+3+i < n)
			    	board[x+3+i][n-y-1] = base[x][y%4]+ temp*12 + 3*n*(stride+1);
			}
		    }
		    else
		    {
			for(int i = 0 ; i < n; i+= 6) // parallel here
			{
			    int stride = i/3;
			    if(n % 4 == 0)
			    {
			    	board[x+i][y] = base8[x][y-blockOfFour(n)]+BaseOfFour*12 + 3*n*stride;
				board[x+3+i][n-y-1] = base8[x][y-blockOfFour(n)]+ BaseOfFour*12 + 3*n*(stride+1); //using base 8
			    }
 			    if(n % 4 == 1)
			    {
			    	board[x+i][y] = base13[x][y-blockOfFour(n)]+ BaseOfFour * 12 + 3*n*stride;
				if(x+3+i < n)
				   board[x+3+i][n-y-1] = base13[x][y-blockOfFour(n)]+ BaseOfFour * 12 + 3*n*(stride+1); //using base 13
			    }
			    if(n % 4 == 2)
			    {
			    	board[x+i][y] = base14[x][y-blockOfFour(n)]+ BaseOfFour * 12 + 3*n*stride;
				if(x+3+i < n)
				   board[x+3+i][n-y-1] = base14[x][y-blockOfFour(n)]+ BaseOfFour * 12 + 3*n*(stride+1); // using base 14
			    }
			    if(n % 4 == 3)
			    {
			    	board[x+i][y] = base7[x][y-blockOfFour(n)]+ BaseOfFour * 12 + 3*n*stride;
				if(x+3+i < n)
				   board[x+3+i][n-y-1] = base7[x][y-blockOfFour(n)]+ BaseOfFour * 12 + 3*n*(stride+1); // using base 7
			    }
			}
		    }
	       }   
	   }
	   break;
	case 1:
	   for (int x = 0; x < 3; x++)
	   {
	       for(int y = 0; y < n; y++)
	       {
		    if(y < blockOfFour(n))
		    {
			int temp = y/4; // temp get index of blockOf4
			for(int i = 0; i < n-7; i+= 6) // 
			{
			    int stride = i/3; 	    	
			    board[x+i][y] = base[x][y%4]+ temp*12 + 3*n*stride; 
			    if(x+3+i < n-7)
			    	board[x+3+i][n-y-1] = base[x][y%4]+ temp*12 + 3*n*(stride+1);
			}
		    }// end of if
		    else
		    {
			for(int i = 0 ; i < n-7; i+= 6) // parallel here
			{
			    int stride = i/3;
			    if(n % 4 == 0)
			    {
			    	board[x+i][y] = base8[x][y-blockOfFour(n)]+BaseOfFour*12 + 3*n*stride;
				if(x+3+i < n-7)// Don't want to get in the 7 stride
				board[x+3+i][n-y-1] = base8[x][y-blockOfFour(n)]+ BaseOfFour*12 + 3*n*(stride+1);
			    }
 			    if(n % 4 == 1)
			    {
			    	board[x+i][y] = base13[x][y-blockOfFour(n)]+ BaseOfFour * 12 + 3*n*stride;
				if(x+3+i < n-7) // Don't want to get in the 7 stride
				   board[x+3+i][n-y-1] = base13[x][y-blockOfFour(n)]+ BaseOfFour * 12 + 3*n*(stride+1);
			    }
			    if(n % 4 == 2)
			    {
			    	board[x+i][y] = base14[x][y-blockOfFour(n)]+ BaseOfFour * 12 + 3*n*stride;
				if(x+3+i < n-7) // Don't want to get in the 7 stride
				   board[x+3+i][n-y-1] = base14[x][y-blockOfFour(n)]+ BaseOfFour * 12 + 3*n*(stride+1);
			    }
			    if(n % 4 == 3)
			    {
			    	board[x+i][y] = base7[x][y-blockOfFour(n)]+ BaseOfFour * 12 + 3*n*stride;
				if(x+3+i < n-7) // Don't want to get in the 7 stride
				   board[x+3+i][n-y-1] = base7[x][y-blockOfFour(n)]+ BaseOfFour * 12 + 3*n*(stride+1);
			    }
			}
		    }// end of else
	       }// end of for y   
	   }// end of big 4
	   //Handling 7*n here
	   for(int x = 0; x < 7; x++)
	   {
		for(int y = 0; y < n-7 ; y++) // NOTE !!!!
		{
		    if(y%6 == 0) //0,6,12....
		    {
			int temp = y/6;
		    	board[n+x-7][y] = tranposeBase7[x][y%3]+ 2*temp*21;
			board[n+x-7][y+1] = tranposeBase7[x][y%3+1]+ 2*temp*21;
			board[n+x-7][y+2] = tranposeBase7[x][y%3+2]+ 2*temp*21;
		    }
		    else if( y%3 == 0) //3,9,15,21...
		    {
			int temp = y/6;
		    	board[n-x-1][y] = tranposeBase7[x][y%3]+ 21 + 2*temp*21;
			board[n-x-1][y+1] = tranposeBase7[x][y%3+1]+ 21 + 2*temp*21;
			board[n-x-1][y+2] = tranposeBase7[x][y%3+2]+ 21 + 2*temp*21;
		    }
		    /*int temp = y/3; // getting index of block of 7*3
		    for(int i = 0; i < n-7; i+= 3) // 
			{  
			    int stride = i/3; 	    	
			    board[n+x-7][y] = tranposeBase7[x][y%3] + temp*21 + 3*n*stride;
			    if(x+3+i < n-7)
			    	board[x+3+i][n-y-1] = base[x][y%4]+ temp*12 + 3*n*(stride+1);
			}*/
		}
	   }
	
	break;
      case 2:
	 for (int x = 0; x < 3; x++)
	   {
	       for(int y = 0; y < n; y++)
	       {
		    if(y < blockOfFour(n))
		    {
			int temp = y/4;	
			for(int i = 0; i < n-8; i+= 6) 
			{
			    int stride = i/3; 	    	
			    board[x+i][y] = base[x][y%4]+ temp*12 + 3*n*stride;
			    if(x+3+i < n-8)
			    	board[x+3+i][n-y-1] = base[x][y%4]+ temp*12 + 3*n*(stride+1);
			}
		    }// end of if
		    else
		    {
			for(int i = 0 ; i < n-8; i+= 6)
			{
			    int stride = i/3;
			    if(n % 4 == 0)
			    {
			    	board[x+i][y] = base8[x][y-blockOfFour(n)]+BaseOfFour*12 + 3*n*stride;
				if(x+3+i < n-8)// Don't want to get in the 8 stride
				board[x+3+i][n-y-1] = base8[x][y-blockOfFour(n)]+ BaseOfFour*12 + 3*n*(stride+1);
			    }
 			    if(n % 4 == 1)
			    {
			    	board[x+i][y] = base13[x][y-blockOfFour(n)]+ BaseOfFour * 12 + 3*n*stride;
				if(x+3+i < n-8) // Don't want to get in the 8 stride
				   board[x+3+i][n-y-1] = base13[x][y-blockOfFour(n)]+ BaseOfFour * 12 + 3*n*(stride+1);
			    }
			    if(n % 4 == 2)
			    {
			    	board[x+i][y] = base14[x][y-blockOfFour(n)]+ BaseOfFour * 12 + 3*n*stride;
				if(x+3+i < n-8) // Don't want to get in the 8 stride
				   board[x+3+i][n-y-1] = base14[x][y-blockOfFour(n)]+ BaseOfFour * 12 + 3*n*(stride+1);
			    }
			    if(n % 4 == 3)
			    {
			    	board[x+i][y] = base7[x][y-blockOfFour(n)]+ BaseOfFour * 12 + 3*n*stride;
				if(x+3+i < n-8) // Don't want to get in the 8 stride
				   board[x+3+i][n-y-1] = base7[x][y-blockOfFour(n)]+ BaseOfFour * 12 + 3*n*(stride+1);
			    }
			}
		    }// end of else
	       }// end of for y   
	   }// end of big 4
	// Handling 8*n stride here
	


    } // end of switch
    for (int x = 0; x < n; x++) {
        for (int y = 0; y < n; y++)
            cout << board[x][y]<< "\t";
        cout << endl;
    }
}

int main()
{
    loadData();
    int n;
    cout << "Enter size of board:";
    cin >> n;
    solveBoard(n);

    /*for(int x = 0; x < 3; x++)
    {
	for(int y = 0; y < 7 ; y++)
	    	cout << base7[x][y] << "\t";                
	cout << endl;
    }
    for(int x = 0; x < 7; x++)
    {
	for(int y = 0; y < 3 ; y++)
	    	cout << tranposeBase7[x][y] << "\t";   
	cout << endl;             
    }*/
    return 0;
}
